#include <hip/hip_runtime.h>
#include <cstdio>

/*
Adapted from
https://wagonhelm.github.io/articles/2018-03/detecting-cuda-capability-with-cmake
*/
int main() {
  hipDeviceProp_t dP;
  int err = hipGetDeviceProperties(&dP, 0);
  if(err != hipSuccess) {
      hipError_t error = hipGetLastError();
      printf("CUDA error: %s", hipGetErrorString(error));
      return err; /* Failure */
  }
  printf("-arch=sm_%d%d", dP.major, dP.minor);
  return 0; /* Success */
}
