#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include "rippleKernel.h"

__global__ void kernel(color3 *ptr, int w, int h, int ticks);

void RippleKernel::update(ImageBuffer* img) {
  int tdim = 8; // number of threads in x/y direction per block

  int w = img->width;
  int h = img->height;

  /* set up grid dimension */
  dim3 blocks((w+(tdim-1)) / tdim, (h+(tdim-1)) / tdim);
  /* set up block dimension */
  dim3 threads_block(tdim, tdim);

  /* call the CUDA kernel with grid dimension */
  kernel<<<blocks, threads_block>>>(img->buffer, w, h, m_ticks);

  /* step size controls speed of animation */
  m_ticks += 2;
}

__global__ void kernel(color3 *ptr, int w, int h, int ticks) {
  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * w;

  // compute distance from center of image
  float fx = x - w / 2;
  float fy = y - h / 2;
  float d = sqrtf(fx * fx + fy * fy);

  // use distance to modulate grey value intensity
  unsigned char grey = (unsigned char)(128.0f +
                                       127.0f * cos(d / 10.0f - ticks / 7.0f) /
                                           (d / 10.0f + 1.0f));
  if(x<w && y<h){
  ptr[offset].r = grey;
  ptr[offset].g = grey;
  ptr[offset].b = grey;
  }
}
