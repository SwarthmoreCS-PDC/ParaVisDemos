#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <unistd.h>
#include "imageFilter.h"

__global__ void kernel(color3 *ptr, int w, int h, int ticks);

void ImageFilter::update(ImageBuffer* img) {
  int tdim = 8; // number of threads in x/y direction per block

  int w = img->width;
  int h = img->height;

  /* set up grid dimension */
  dim3 blocks((w+(tdim-1)) / tdim, (h+(tdim-1)) / tdim);
  /* set up block dimension */
  dim3 threads_block(tdim, tdim);

  /* call the CUDA kernel with grid dimension */
  kernel<<<blocks, threads_block>>>(img->buffer, w, h, m_ticks);
  usleep(500000);
  printf("%d\n",m_ticks);
  /* step size controls speed of animation */
  m_ticks += 1;
}

__global__ void kernel(color3 *ptr, int w, int h, int ticks) {
  // map from threadIdx/BlockIdx to pixel position
  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * w;

  // compute distance from center of image
  float fx = x - w / 2;
  float fy = y - h / 2;
  float d = sqrtf(fx * fx + fy * fy);

  // use distance to modulate grey value intensity
  int grey = (int) (20.0f * cos(d / 10.0f - ticks / 7.0f) / (d / 10.0f + 1.0f));

  if(x<w && y<h){
  if(ptr[offset].r<200){
    ptr[offset].r += 10;
  }
  if(ptr[offset].r>210){
    ptr[offset].r -= 60;
  }
  if(ptr[offset].b<200){
    ptr[offset].b += 5;
  }
  if(ptr[offset].b>210){
    ptr[offset].b += 70;
  }
  }
}
